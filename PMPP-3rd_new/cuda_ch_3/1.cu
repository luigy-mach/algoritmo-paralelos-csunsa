#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define T 2 // max threads x bloque
#define N 5  




__global__ void sumaMatrices(int *m1, int *m2, int *m3) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int fil = blockIdx.y * blockDim.y + threadIdx.y;

  int indice = fil * N + col;
  if (col < N && fil < N) {
    m3[indice] = m1[indice] + m2[indice];
  }

}


void fill(int m[N][N], int n){
  int i,j;
  int c;
   for (i = 0; i < N; i++) {
    c = rand()%99;
    //c = 0;
    for (j = 0; j < N; j++) {
      m[i][j] = c;
      c++;
    }
  }    
}


void print(int m[N][N], int n){
  printf("------------------------------------\n"); 
  int i,j;
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      printf(" %d ", m[i][j]);
     }
    printf("\n\n"); 
  }
  printf("------------------------------------\n"); 
}



int main(int argc, char** argv) {

  int m1[N][N];
  int m2[N][N];
  int m3[N][N];

  fill(m1,N);
  fill(m2,N);


  int *dm1, *dm2, *dm3;

  hipMalloc((void**) &dm1, N * N * sizeof(int));
  hipMalloc((void**) &dm2, N * N * sizeof(int));
  hipMalloc((void**) &dm3, N * N * sizeof(int));

  hipMemcpy(dm1, m1, N * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dm2, m2, N * N * sizeof(int), hipMemcpyHostToDevice);

  int B = (int) ceil((float) N / (float) T);

  dim3 dimThreadsBloque(T, T);
  dim3 dimBloques(B, B);

  sumaMatrices<<<dimBloques, dimThreadsBloque>>>(dm1, dm2, dm3);

  hipMemcpy(m3, dm3, N * N * sizeof(int), hipMemcpyDeviceToHost);
  //cudaMemcpy(m2, dm2, N * N * sizeof(int), cudaMemcpyDeviceToHost);

 

  print(m1,N);
  print(m2,N);
  print(m3,N);

  printf("\nB = %d", B);
  printf("\n%d, %d",dimBloques.x, dimBloques.y);
  printf("\n%d, %d\n",dimThreadsBloque.x, dimThreadsBloque.y);


  hipFree(dm1);
  hipFree(dm2);
  hipFree(dm3);
  

 	return 0;
}
