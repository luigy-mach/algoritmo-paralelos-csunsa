#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define T 2 // max threads x bloque
#define N 6

#define CHANNELS 3


__global__ 
void colorConvert(int * grayImage, int * rgbImage, int width, int height){
    //__syncthreads();

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
   
    if (x < width && y < height) {
      // get 1D coordinate for the grayscale image
      int grayOffset = y*(width) + x;
      // one can think of the RGB image having
      // CHANNEL times columns than the gray scale image
      int rgbOffset = grayOffset ;
        int r = rgbImage[rgbOffset + 0]; // red value for pixel
        int g = rgbImage[rgbOffset + 1]; // green value for pixel
        int b = rgbImage[rgbOffset + 2]; // blue value for pixel
      // perform the rescaling and store it
      // We multiply by floating point constants
      grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
      //grayImage[grayOffset] = 0;
    }
}


void fill(int m[N][N], int n){
  int i,j;
   for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      m[i][j] = -1;
    }
  }    
}

void fill(int m[N][N*CHANNELS]){
  int i,j;
   for (i = 0; i < N; i++) {
    for (j = 0; j < N*CHANNELS; j++) {
      m[i][j] = rand()%9;
    }
  }    
}


void print(int m[N][N], int n){
  printf("------------------------------------\n"); 
  int i,j;
  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      printf(" %d ", m[i][j]);
     }
    printf("\n\n"); 
  }
  printf("------------------------------------\n"); 
}


void print(int m[N][N*CHANNELS]){
  printf("------------------------------------\n"); 
  int i,j;
  for (i = 0; i < N; i++) {
    for (j = 0; j < N*CHANNELS; j++) {
      printf(" %d ", m[i][j]);
      if(((j+1)%CHANNELS)==0){
        printf(" | ");
      }
     }
    printf("\n\n"); 
  }
  printf("------------------------------------\n"); 
}



int main(int argc, char** argv) {

  int m1[N][N*CHANNELS];
  int m2[N][N];

  fill(m1);
  fill(m2,N);


  int *dm1, *dm2;

  hipMalloc((void**) &dm1, N * CHANNELS * N * sizeof(int));
  hipMalloc((void**) &dm2, N * N * sizeof(int));

  hipMemcpy(dm1, m1, N * CHANNELS * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dm2, m2, N * N * sizeof(int), hipMemcpyHostToDevice);

  int B = (int) ceil((float) N / (float) T);

  dim3 dimBloques(B, B);
  dim3 dimThreadsBloque(T, T);

  int w=N;
  int h=N;
  colorConvert<<<dimBloques, dimThreadsBloque>>>(dm2, dm1, w, h);


  hipMemcpy(m2, dm2, N * N * sizeof(int), hipMemcpyDeviceToHost);

  printf("&&&&&&&&&&&&&&&&&&\n");
    print(m1);
    print(m2,N);
  printf("&&&&&&&&&&&&&&&&&&\n");

  hipFree(dm1);
  hipFree(dm2);  

 	return 0;
}
