#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define T 2 // max threads x bloque

#define N 5  

#define BLUR_SIZE 1


__global__
   void blurKernel(int * in, int * out, int w, int h) {
     int Col = blockIdx.x * blockDim.x + threadIdx.x;
     int Row = blockIdx.y * blockDim.y + threadIdx.y;
     if (Col < w && Row < h) {
       int pixVal = 0;
       int pixels = 0;
       // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
       for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
         for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
           int curRow = Row + blurRow;
           int curCol = Col + blurCol;
           // Verify we have a valid image pixel
           if(curRow > -1 && curRow < h && curCol > -1 && curCol < w){
             pixVal += in[curRow * w + curCol];
             pixels++; // Keep track of number of pixels in the accumulated total
           }
         }
       }
       // Write our new pixel value out
       out[Row * w + Col] = (int)(pixVal / pixels);
     }
 }



void fill(int m[N][N], int n){
  int i,j;
  int c;
   for (i = 0; i < N; i++) {
    c = rand()%99;
    //c = 0;
    for (j = 0; j < N; j++) {
      m[i][j] = c;
      c++;
    }
  }    
}


void print(int m[N][N], int n){
  printf("------------------------------------\n"); 
  int i,j;
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      printf(" %d ", m[i][j]);
     }
    printf("\n\n"); 
  }
  printf("------------------------------------\n"); 
}



int main(int argc, char** argv) {

  int m1[N][N];
  int m2[N][N];

  fill(m1,N);
  fill(m2,N);

  int *dm1, *dm2;

  hipMalloc((void**) &dm1, N * N * sizeof(int));
  hipMalloc((void**) &dm2, N * N * sizeof(int));

  hipMemcpy(dm1, m1, N * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dm2, m2, N * N * sizeof(int), hipMemcpyHostToDevice);

  int B = (int) ceil((float) N / (float) T);

  dim3 dimBloques(B, B);
  dim3 dimThreadsBloque(T, T);

  int w=N;
  int h=N;

  blurKernel<<<dimBloques, dimThreadsBloque>>>(dm1, dm2, w, h);

  hipMemcpy(m2, dm2, N * N * sizeof(int), hipMemcpyDeviceToHost);

  printf("&&&&&&&&&&&&&&&&&&\n");
  print(m1,N);
  print(m2,N);
  printf("&&&&&&&&&&&&&&&&&&\n");

  hipFree(dm1);
  hipFree(dm2);  

 	return 0;
}
