
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10
#define T 2



__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
}


void fill(int mat[N][N]){
	int i,j;
	for(i=0;i<N;i++)
		for(j=0;j<N;j++)
			mat[i][j]=rand()%5;

}


void fill_zero(int mat[N][N]){
	int i,j;
	for(i=0;i<N;i++)
		for(j=0;j<N;j++)
			mat[i][j]=0;

}


void print(int mat[N][N]){
	int i,j;
	for(i=0;i<N;i++){
		for(j=0;j<N;j++)
			printf("%i",mat[i][j]);
		printf("\n");	
	}

}

int main(){

	int A[N][N];
	int B[N][N];
	int C[N][N];

	fill(A); fill(B); fill_zero(C);

	printf("//////////////////////\n");
	print(A);
	printf("//////////////////////\n");
	print(B);
	printf("//////////////////////\n");
	print(C);
	printf("//////////////////////\n");

	int (*pA)[N], (*pB)[N], (*pC)[N];

	hipMalloc((void**)&pA, (N*N)*sizeof(int));
	hipMalloc((void**)&pB, (N*N)*sizeof(int));
	hipMalloc((void**)&pC, (N*N)*sizeof(int));
	 
	hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(N,N);
	MatAdd<<<numBlocks,threadsPerBlock>>>(pA,pB,pC);

	hipMemcpy(C, pC, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	printf("//////////////////////\n");
	print(C);
	printf("//////////////////////\n");



	hipFree(pA); 
	hipFree(pB); 
	hipFree(pC);

	printf("\n");

	return 0;
}
