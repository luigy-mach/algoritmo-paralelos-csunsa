#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "qdbmp.h"



#define T 2 // max threads x bloque

#define N 295
#define W 295
#define H 250

#define CHANNELS 3


__global__ 
void colorConvert(int * grayImage, int * rgbImage, int width, int height){
    //__syncthreads();

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
   
    if (x < width && y < height) {
      // get 1D coordinate for the grayscale image
      int grayOffset = y*(width) + x;
      // one can think of the RGB image having
      // CHANNEL times columns than the gray scale image
      int rgbOffset = grayOffset ;
        int r = rgbImage[rgbOffset + 0]; // red value for pixel
        int g = rgbImage[rgbOffset + 1]; // green value for pixel
        int b = rgbImage[rgbOffset + 2]; // blue value for pixel
      // perform the rescaling and store it
      // We multiply by floating point constants
      grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
      //grayImage[grayOffset] = 0;
    }
}


void fill(int m[N][N], int n, int m){
  int i,j;
   for (i = 0; i < n; i++) {
    for (j = 0; j < m; j++) {
      m[i][j] = -1;
    }
  }    
}

void fill(int m[N][N*CHANNELS]){
  int i,j;
   for (i = 0; i < N; i++) {
    for (j = 0; j < N*CHANNELS; j++) {
      m[i][j] = rand()%9;
    }
  }    
}


void print(int m[N][N], int n){
  printf("------------------------------------\n"); 
  int i,j;
  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      printf(" %d ", m[i][j]);
     }
    printf("\n\n"); 
  }
  printf("------------------------------------\n"); 
}


void print(int m[N][N*CHANNELS]){
  printf("------------------------------------\n"); 
  int i,j;
  for (i = 0; i < N; i++) {
    for (j = 0; j < N*CHANNELS; j++) {
      printf(" %d ", m[i][j]);
      if(((j+1)%CHANNELS)==0){
        printf(" | ");
      }
     }
    printf("\n\n"); 
  }
  printf("------------------------------------\n"); 
}



int main(int argc, char** argv) {

  int m1[H][W*CHANNELS];
  int m2[H][W];

  fill(m1,H,W*CHANNELS);
  fill(m2,H,W);


  printf("///////////////////////////////////////////////////////////////////////\n");
  printf("///////////////////////////////////////////////////////////////////////\n");
  UCHAR r, g, b;
  UINT  width, height;
  UINT  x, y;
  BMP*  bmp;

  /* Check arguments */
  if ( argc != 3 )
  {
    fprintf( stderr, "Usage: %s <input file> <output file>\n", argv[ 0 ] );
    return 0;
  }

  printf("///////////////////////////////////////////////////////////////////////\n");
  /* Read an image file */
  bmp = BMP_ReadFile( argv[ 1 ] );
  BMP_CHECK_ERROR( stdout, -1 );

  /* Get image's dimensions */
  width = BMP_GetWidth( bmp );
  height = BMP_GetHeight( bmp );



|/* Iterate through all the image's pixels */
  for ( x = 0 ; x < width ; ++x )
  {
    for ( y = 0 ; y < height ; ++y )
    {
      /* Get pixel's RGB values */
      BMP_GetPixelRGB( bmp, x, y, &r, &g, &b );


      
    }
  }




  printf("///////////////////////////////////////////////////////////////////////\n");
  printf("///////////////////////////////////////////////////////////////////////\n");






  int *dm1, *dm2;

  hipMalloc((void**) &dm1, H * CHANNELS * W * sizeof(int));
  hipMalloc((void**) &dm2, H * W * sizeof(int));

  hipMemcpy(dm1, m1, H * CHANNELS * W * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dm2, m2, H * W * sizeof(int), hipMemcpyHostToDevice);

  int B = (int) ceil((float) N / (float) T);

  dim3 dimBloques(B, B);
  dim3 dimThreadsBloque(T, T);

  int w=W;
  int h=H;
  colorConvert<<<dimBloques, dimThreadsBloque>>>(dm2, dm1, w, h);


  hipMemcpy(m2, dm2, N * N * sizeof(int), hipMemcpyDeviceToHost);

  printf("&&&&&&&&&&&&&&&&&&\n");
    print(m1);
    print(m2,N);
  printf("&&&&&&&&&&&&&&&&&&\n");


  printf("///////////////////////////////////////////////////////////////////////\n");
  printf("///////////////////////////////////////////////////////////////////////\n");


  /* Iterate through all the image's pixels */
  for ( x = 0 ; x < width ; ++x )
  {
    for ( y = 0 ; y < height ; ++y )
    {
      BMP_SetPixelRGB( bmp, x, y, m2[i][j], 0, 0 );
    }
  }

  printf("///////////////////////////////////////////////////////////////////////\n");



  /* Save result */
  BMP_WriteFile( bmp, argv[ 2 ] );
  BMP_CHECK_ERROR( stdout, -2 );


  /* Free all memory allocated for the image */
  BMP_Free( bmp );


  printf("///////////////////////////////////////////////////////////////////////\n");

  hipFree(dm1);
  hipFree(dm2);  



 	return 0;
}
