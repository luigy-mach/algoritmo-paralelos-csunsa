
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TAM 3

#define N 2
#define T 6
#define TPB 16


__global__ 
void sum_matrix(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){ // revisar <<<<<<<<<<<<<<<<<<<<<<<<<<<<
		//*(*(dd_mat_a+y)+x)=-9;
		*(*(dd_mat_c+y)+x)= *(*(dd_mat_a+y)+x) + *(*(dd_mat_b+y)+x);
	}
}




void create3(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}




void fill(int** mat, int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%2;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}




void create5(int**& mat, int**& d_mat, int**& dd_mat, int n, int m, int fillValue=-1){
	int i;

	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	for(i=1;i<n;i++){
		mat[i] = mat[i-1]+m;
	}

	if(fillValue==-1){
		fill(mat,n,m);	
	}
	else{
		fill_value(mat,n,m,fillValue);
	}

	
	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;



	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * m * n );
	hipMemcpy(d_mat[0], mat[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat[i]=(d_mat[i-1]+m);
	}	
	
	hipMalloc((void***)&dd_mat,size_row);
	hipMemcpy(dd_mat,d_mat,size_row,hipMemcpyHostToDevice);
	

}




int main(){

	if(N*T<TAM){
		printf("no cubre la matriz\n");
		return 0;
	}

	int n = TAM;
	int m = TAM;

	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	create5(mat_a,d_mat_a,dd_mat_a,n,m);
	create5(mat_b,d_mat_b,dd_mat_b,n,m);
	create5(mat_c,d_mat_c,dd_mat_c,n,m,0);

	int i;

	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

/*	
	int** mat_a;

	create3(&mat_a,n,m);

	fill(mat_a,n,m);

	int **d_mat_a;
	int **dd_mat_a;
		

	d_mat_a = (int**) malloc(size_row);
	cudaMalloc((void**)& d_mat_a[0], sizeof(int) * m * n );
	cudaMemcpy(d_mat_a[0], mat_a[0], sizeof(int) * m * n ,cudaMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat_a[i]=(d_mat_a[i-1]+m);
	}	
	
	cudaMalloc((void***)&dd_mat_a,size_row);
	cudaMemcpy(dd_mat_a,d_mat_a,size_row,cudaMemcpyHostToDevice);
*/
	print(mat_a,n,m);
	printf("//////////////////\n");
	print(mat_b,n,m);
	printf("//////////////////\n");
	print(mat_c,n,m);
	printf("//////////////////\n");
	printf("//////////////////\n");



	dim3 blockNum(TPB,TPB,1);
	dim3 grid((blockNum.x-1+n)/blockNum.x,(blockNum.y-1+m)/blockNum.y,1);

	sum_matrix<<<grid,blockNum>>>(dd_mat_a,dd_mat_b,dd_mat_c,n,m);

	for(i=0;i<n;i++){
		hipMemcpy(mat_c[i],d_mat_c[i],size_col,hipMemcpyDeviceToHost);	
	}	

	printf("//////////////////\n");
	printf("//////////////////\n");
	//print(mat_a,n,m);
	printf("//////////////////\n");
	//print(mat_b,n,m);
	printf("//////////////////\n");
	print(mat_c,n,m);


	return 0;
}