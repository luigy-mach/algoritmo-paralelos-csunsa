
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 5	
#define T 2


__global__ 
void sum_matrix(int** dd_mat_a, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){
		*(*(dd_mat_a+y)+x)=-9;
	}
}




void create3(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}




void fill(int** mat, int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%2;
	}
}


void fill_zero(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}



int main(){

	int n = N;
	int m = N+4;

	int** mat_a;

	create3(&mat_a,n,m);

	fill(mat_a,n,m);

	int **d_mat_a;
	int **dd_mat_a;
		

	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

	int i;

	d_mat_a = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat_a[0], sizeof(int) * m * n );
	hipMemcpy(d_mat_a[0], mat_a[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
	//for(i=0;i<n;i++){
		d_mat_a[i]=(d_mat_a[i-1]+m);
		//d_mat_a[i]=(d_mat_a[i]+m);
	}	
	
	hipMalloc((void***)&dd_mat_a,size_row);
	hipMemcpy(dd_mat_a,d_mat_a,size_row,hipMemcpyHostToDevice);


	
	print(mat_a,n,m);
	printf("//////////////////\n");

	dim3 grid(ceil(N/T)+1,ceil(N/T)+1,1);
	dim3 blockNum(T,T,1);

	sum_matrix<<<grid,blockNum>>>(dd_mat_a,n,m);
	
	//cudaMemcpy(mat_a,*d_mat_a,size_row,cudaMemcpyDeviceToHost);
	for(i=0;i<n;i++){
		hipMemcpy(mat_a[i],d_mat_a[i],size_col,hipMemcpyDeviceToHost);	
	}	

	
	printf("//////////////////\n");
	print(mat_a,n,m);


	return 0;
}