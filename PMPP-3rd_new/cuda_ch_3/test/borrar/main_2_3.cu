
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 4	
#define T 2



__global__ void sum_matrix(int**&d_mat_a, int**&dd_mat_a, int n, int m){
  
				//int x = threadIdx.x + blockIdx.x*blockDim.x;
				//int y = threadIdx.y + blockIdx.y*blockDim.y;
				//int* temp;
				//temp = dd_mat_a[0];
				//temp[2]=-1;

				d_mat_a[0][0] = -1;
				d_mat_a[0][1] = -2;
				d_mat_a[0][2] = -3;

				dd_mat_a[0][-1] = -4;
				//dd_mat_a[0][-2] = -5;
				//dd_mat_a[0][-3] = -6;
				
}

void create(int**&mat,int n){
	mat = (int **)malloc(sizeof(int*)*n);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (int*)malloc(sizeof(int)*n);
	}
}

void create2(int** & mat,int n, int m){
	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<=n;i++){
		mat[i] = (*mat+i*m);
	}
}

void fill(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			mat[i][j] = rand()%10;
	}
}

void fill_zero(int** mat,int n, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<n;j++)
			mat[i][j] = value;
}



void print(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}

/*
 void create_matrix(int**&mat, int**&h_mat, int**&d_mat, int n, int m){
	int size_n=sizeof(int)*n;
	int size_m=sizeof(int)*m;

	h_mat = (int**)malloc(size_n);

	int i;
	for(i=0;i<n;i++){
		printf(">>>>>\n");
		cudaMalloc((void**)& h_mat[i],size_n);
		cudaMemcpy(h_mat[i],&mat[i][0],size_m,cudaMemcpyHostToDevice);
	}

	cudaMalloc((void*** )& d_mat,size_n);
	cudaMemcpy(d_mat,h_mat,size_n,cudaMemcpyHostToDevice);

}*/






int main(){

	int   n = N;
	int   m = N;

	int** h_mat_a;
		create2(h_mat_a,n,m);
		fill_zero(h_mat_a,n);
		print(h_mat_a,n);
	printf("//////////////////\n");



	int ** d_mat_a;	int ** dd_mat_a; 

	
	int i;
	 
	///////////////////////

    d_mat_a = (int** )malloc(sizeof(int*)*n);
    hipMalloc((void** )& d_mat_a[0], n*sizeof(int)*m);
    hipMemcpy(d_mat_a[0],&h_mat_a[0][0], n*sizeof(int)*m,hipMemcpyHostToDevice);
    for(i=1;i<n;i++){
    	printf(">wwww>>>>\n");
    	d_mat_a[i]  = &(d_mat_a[0][i*m]);
    	//d_mat_a[i]  = (*d_mat_a+i*m);
    }
    	
    hipMalloc((void*** )& dd_mat_a,sizeof(int*)*n);
    hipMemcpy(dd_mat_a,d_mat_a,sizeof(int*)*n,hipMemcpyHostToDevice);    
 
    ///

   
    //////////////////////////

	dim3 grid(ceil(N/T),ceil(N/T),1);
	dim3 blockNum(T,T,1);

	sum_matrix<<<grid,blockNum>>>(d_mat_a,dd_mat_a,n,m);

	hipMemcpy(h_mat_a[0],&d_mat_a[0][0],n*sizeof(int)*m,hipMemcpyDeviceToHost);	
	
   	printf(">copiado>>>>\n");

	printf("///////CCCCCC///////////\n");
	print(h_mat_a,n);


	return 0;
}