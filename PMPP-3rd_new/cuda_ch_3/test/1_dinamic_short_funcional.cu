
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 4	
#define T 2


__global__ void sum_matrix(int** d_mat_a, int** dd_mat_a, int n, int m, int size){
				dd_mat_a[0][3]=-2;
				dd_mat_a[1][3]=-2;
				dd_mat_a[2][3]=-2;
				dd_mat_a[3][3]=-2;
				//**dd_mat_a=-1;
}

/*
void create(int** & mat,int n){
	mat = (int **)malloc(sizeof(int*)*n);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (int*)malloc(sizeof(int)*n);
	}
}

void create2(int**& mat,int n, int m){
	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	//mat[1] = mat[0]+1*m;
	for(i=1;i<n;i++){
		mat[i] = mat[0]+i*m;
	}
}*/


void create3(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}



/*
void create4(int** mat,int n, int m){
	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		mat[i] = mat[0]+i*m;
	}
}
*/


void fill(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			mat[i][j] = rand()%10;
	}
}


void fill_zero(int** mat,int n, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<n;j++)
			mat[i][j] = value;
}


void print(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}



int main(){

	int n = N;
	int m = N;

	int** mat_a;

	create3(&mat_a,n,m);
	//create4(mat_a,n,m);

	fill(mat_a,n);

	int **d_mat_a;
	int **dd_mat_a;
	int size = sizeof(int*) * n * sizeof(int) * n;
	
	int i;

	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

	d_mat_a = (int**) malloc(size_row);
	for(i=0;i<n;i++){
		hipMalloc((void**)&d_mat_a[i],size_col);
	}
	for(i=0;i<n;i++){
		hipMemcpy(d_mat_a[i],mat_a[i],size_col,hipMemcpyHostToDevice);	
	}	

	hipMalloc((void***)&dd_mat_a,size_row);
	hipMemcpy(dd_mat_a,d_mat_a,size_row,hipMemcpyHostToDevice);

	
	print(mat_a,n);
	printf("//////////////////\n");

	//dim3 grid(ceil(N/T),ceil(N/T),1);
	//dim3 blockNum(T,T,1);
	dim3 grid(T,T,1);
	dim3 blockNum(n*T,m*T,1);

	sum_matrix<<<grid,blockNum>>>(d_mat_a,dd_mat_a,n,n,size);
	
	//cudaMemcpy(mat_a,*d_mat_a,size_row,cudaMemcpyDeviceToHost);
	for(i=0;i<n;i++){
		hipMemcpy(mat_a[i],d_mat_a[i],size_col,hipMemcpyDeviceToHost);	
	}	

	
	printf("//////////////////\n");
	print(mat_a,n);


	return 0;
}