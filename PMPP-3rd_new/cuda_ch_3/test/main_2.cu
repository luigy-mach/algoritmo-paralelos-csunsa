
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 4	
#define T 2


__global__ void sum_matrix(int** mat1, int** ddmat1, int** mat2, int** ddmat2, int** mat3, int** ddmat3, int n, int m){
				int x = threadIdx.x + blockIdx.x*blockDim.x;
				int y = threadIdx.y + blockIdx.y*blockDim.y;
				mat3[1][x] = 0;
				
}

void create(int**&mat,int n){
	mat = (int **)malloc(sizeof(int*)*n);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (int*)malloc(sizeof(int)*n);
	}
}

void create2(int** & mat,int n, int m){
	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (*mat+i*m);
	}
}

void fill(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			mat[i][j] = rand()%10;
	}
}

void fill_zero(int** mat,int n, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<n;j++)
			mat[i][j] = value;
}



void print(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}

/*
 void create_matrix(int**&mat, int**&h_mat, int**&d_mat, int n, int m){
	int size_n=sizeof(int)*n;
	int size_m=sizeof(int)*m;

	h_mat = (int**)malloc(size_n);

	int i;
	for(i=0;i<n;i++){
		printf(">>>>>\n");
		cudaMalloc((void**)& h_mat[i],size_n);
		cudaMemcpy(h_mat[i],&mat[i][0],size_m,cudaMemcpyHostToDevice);
	}

	cudaMalloc((void*** )& d_mat,size_n);
	cudaMemcpy(d_mat,h_mat,size_n,cudaMemcpyHostToDevice);

}*/


int main(){

	int   n = N;
	int   m = N;
	int** mat_a;
		create(mat_a,n);
		fill(mat_a,n);
		print(mat_a,n);
	printf("//////////////////\n");

	int** mat_b;
		create(mat_b,n);
		fill(mat_b,n);
		print(mat_b,n);

	printf("//////////////////\n");
	
	int** mat_c;
		create(mat_c,n);
		fill_zero(mat_c,n,-1);
		print(mat_c,n);

	printf("//////////////////\n");

	int ** h_mat_a;	int ** d_mat_a; int ** dd_mat_a;
	int ** h_mat_b;	int ** d_mat_b; int ** dd_mat_b;
	int ** h_mat_c;	int ** d_mat_c; int ** dd_mat_c;

	
	int i;
	 
	///////////////////////

    h_mat_a = (int** )malloc(sizeof(int*)*n);
    for(i=0;i<n;i++){
    	printf(">>>>>\n");
        hipMalloc((void** )& h_mat_a[i], sizeof(int)*m);
        hipMemcpy(h_mat_a[i],&mat_a[i][0],sizeof(int)*m,hipMemcpyHostToDevice);
    }
    
    hipMalloc((void*** )& d_mat_a,sizeof(int*)*n);
    hipMemcpy(d_mat_a,h_mat_a,sizeof(int)*n,hipMemcpyHostToDevice);    

    hipMalloc((void*** )& dd_mat_a,sizeof(int*)*n);
    hipMemcpy(dd_mat_a,h_mat_a,sizeof(int)*n,hipMemcpyHostToDevice);    
 
    ///

  	h_mat_b = (int** )malloc(sizeof(int*)*n);
    for(i=0;i<n;i++){
    	printf(">>>>>\n");
        hipMalloc((void** )& h_mat_b[i], sizeof(int)*m);
        hipMemcpy(h_mat_b[i],&mat_b[i][0],sizeof(int)*m,hipMemcpyHostToDevice);
    }
    
    hipMalloc((void*** )& d_mat_b,sizeof(int*)*n);
    hipMemcpy(d_mat_b,h_mat_b,sizeof(int)*n,hipMemcpyHostToDevice);    

    hipMalloc((void*** )& dd_mat_b,sizeof(int*)*n);
    hipMemcpy(dd_mat_b,h_mat_b,sizeof(int)*n,hipMemcpyHostToDevice);    

    ///
    
  	h_mat_c = (int** )malloc(sizeof(int*)*n);
    for(i=0;i<n;i++){
    	printf(">>>>>\n");
        hipMalloc((void** )& h_mat_c[i], sizeof(int)*m);
        hipMemcpy(h_mat_c[i],&mat_c[i][0],sizeof(int)*m,hipMemcpyHostToDevice);
    }
    
    hipMalloc((void*** )& d_mat_c,sizeof(int*)*n);
    hipMemcpy(d_mat_c,h_mat_c,sizeof(int)*n,hipMemcpyHostToDevice);    

    hipMalloc((void*** )& dd_mat_c,sizeof(int*)*n);
    hipMemcpy(dd_mat_c,h_mat_c,sizeof(int)*n,hipMemcpyHostToDevice);    

    ////////////////77

	//create_matrix(mat_a,h_mat_a,d_mat_a,n,m);
	//create_matrix(mat_b,h_mat_b,d_mat_b,n,m);
	//create_matrix(mat_c,h_mat_c,d_mat_c,n,m);



	dim3 grid(ceil(N/T),ceil(N/T),1);
	dim3 blockNum(T,T,1);

	//int size = sizeof(int)*n*n;
	sum_matrix<<<grid,blockNum>>>(d_mat_a,dd_mat_a,d_mat_b,dd_mat_b,d_mat_c,dd_mat_c,n,m);

	
	for(i=0;i<n;i++){
		hipMemcpy(&mat_c[i][0],h_mat_c[i],sizeof(int)*m,hipMemcpyDeviceToHost);	
	}
	

	printf("///////CCCCCC///////////\n");
	print(mat_c,n);


	return 0;
}