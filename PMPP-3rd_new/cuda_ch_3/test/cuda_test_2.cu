
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TAM 10

#define N 2
#define T 6




__global__ 
void sum_matrix(int** dd_mat_a, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){ // revisar <<<<<<<<<<<<<<<<<<<<<<<<<<<<
		*(*(dd_mat_a+y)+x)=-3;
		printf("%d",*(*(dd_mat_a+y)+x));
	}

}


__global__ 
void sum_matrix2(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){ 
		*(*(dd_mat_c+y)+x)=*(*(dd_mat_a+y)+x)+*(*(dd_mat_b+y)+x);
	}

}




void create3(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}



void fill(int** mat, int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%2;
	}
}


void fill_zero(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}

void create_matrices(int** &mat_a,int** &d_mat_a,int** &dd_mat_a,int n,int m)
{
	int i;

	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

	create3(&mat_a,n,m);
	fill(mat_a,n,m);

	d_mat_a = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat_a[0], sizeof(int) * m * n );
	hipMemcpy(d_mat_a[0], mat_a[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat_a[i]=(d_mat_a[i-1]+m);
	}

	hipMalloc((void***)&dd_mat_a,size_row);
	hipMemcpy(dd_mat_a,d_mat_a,size_row,hipMemcpyHostToDevice);
	
}



int main(){


	int n = TAM;
	int m = TAM;

	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	int i;
	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

    create_matrices(mat_a,d_mat_a,dd_mat_a,n,m);
    create_matrices(mat_b,d_mat_b,dd_mat_b,n,m);
    create_matrices(mat_c,d_mat_c,dd_mat_c,n,m);

    print(mat_a,n,m);
    printf("//////////////////\n");
    print(mat_b,n,m);
    printf("//////////////////\n");
    print(mat_c,n,m);
    printf("//////////////////\n");
    printf("//////////////////\n");
    printf("//////////////////\n");



	dim3 grid(N,N,1);
	dim3 blockNum(T,T,1);

	sum_matrix2<<<grid,blockNum>>>(dd_mat_a,dd_mat_b,dd_mat_c,n,m);

	for(i=0;i<n;i++){
		hipMemcpy(mat_c[i],d_mat_c[i],size_col,hipMemcpyDeviceToHost);	
	}	

	printf("//////////////////\n");
	print(mat_c,n,m);


	return 0;
}