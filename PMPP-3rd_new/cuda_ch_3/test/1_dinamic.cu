
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 4	
#define T 2


/*
int i = threadIdx.x + blockIdx.x*blockDim.x ;
	int j = threadIdx.y + blockIdx.y*blockDim.y ;
	int index_i=0;
	int index_j=0;

		mat3[0][0]= 1;
		mat3[0][1]= 2;
		mat3[0][2]= 3;

	if( i<m && j<n ){
		index_i=i*m;
		index_j=+j;
		//mat3[index_i][index_j]=mat1[index_i][index_j] + mat2[index_i][index_j];
	}
*/


__global__ void sum_matrix(int** mat1, int** mat2, int** mat3, int n, int m, int size){
				mat3[0][0]=1;
				mat3[0][1]=2;
				mat3[0][2]=3;
}

void create(int** & mat,int n){
	mat = (int **)malloc(sizeof(int*)*n);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (int*)malloc(sizeof(int)*n);
	}
}

void create2(int** & mat,int n, int m){
	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (*mat+i*m);
	}
}

void fill(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			mat[i][j] = rand()%10;
	}
}

void fill_zero(int** mat,int n, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<n;j++)
			mat[i][j] = value;
}



void print(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}



int main(){

	int n=N;

	int** mat_a;
	int** mat_b;
	int** mat_c;

	create(mat_a,n);
	create(mat_b,n);
	create(mat_c,n);

	fill(mat_a,n);
	fill(mat_b,n);
	fill_zero(mat_c,n);

	int **d_mat_a, **d_mat_b, **d_mat_c;
	int size=sizeof(int*)*n*sizeof(int)*n;
	
	int i,j;

	int size_row=sizeof(int*)*n;
	int size_col=sizeof(int)*n;

	hipMalloc((void*** )&d_mat_a,size_row);
	for(i=0;i<n;i++){
		hipMalloc((void**)&d_mat_a[i],size_col);
	}

	hipMalloc((void*** )&d_mat_b,size_row);
	for(i=0;i<n;i++){
		hipMalloc((void**)&d_mat_b[i],size_col);
	}

	hipMalloc((void*** )&d_mat_c,size_row);
	for(i=0;i<n;i++){
		hipMalloc((void**)&d_mat_c[i],size_col);
	}

	hipMemcpy(d_mat_a,mat_a,size_row,hipMemcpyHostToDevice);
	for(i=0;i<n;i++){
		hipMemcpy(d_mat_a[i],mat_a[i],size_col,hipMemcpyHostToDevice);	
	}	
	hipMemcpy(d_mat_b,mat_b,size_row,hipMemcpyHostToDevice);
	for(i=0;i<n;i++){
		hipMemcpy(d_mat_b[i],mat_b[i],size_col,hipMemcpyHostToDevice);	
	}	
	hipMemcpy(d_mat_c,mat_c,size_row,hipMemcpyHostToDevice);
	for(i=0;i<n;i++){
		hipMemcpy(d_mat_c[i],mat_c[i],size_col,hipMemcpyHostToDevice);	
	}	

	
	print(mat_a,n);
	printf("//////////////////\n");
	print(mat_b,n);
	printf("///////////////////\n");
	print(mat_c,n);

	dim3 grid(ceil(N/T),ceil(N/T),1);
	dim3 blockNum(T,T,1);

	sum_matrix<<<grid,blockNum>>>(d_mat_a,d_mat_b,d_mat_c,n,n,size);
	
	hipMemcpy(mat_c,d_mat_c,size_row,hipMemcpyDeviceToHost);
	for(i=0;i<n;i++){
		hipMemcpy(d_mat_c[i],mat_c[i],size_col,hipMemcpyDeviceToHost);	
	}	

	
	printf("//////////////////\n");
	print(mat_c,n);


	return 0;
}