
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TAM 50

#define N 6
#define T 10


__global__ 
void sum_matrix(int** dd_mat_a, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){ // revisar <<<<<<<<<<<<<<<<<<<<<<<<<<<<
		*(*(dd_mat_a+y)+x)=-9;
	}
}




void create3(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}




void fill(int** mat, int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%2;
	}
}


void fill_zero(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}






void create4(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}




int main(){

	if(N*T<TAM){
		printf("no cubre la matriz\n");
		return 0;
	}
	
	int n = TAM;
	int m = TAM;

	int** mat_a;

	create3(&mat_a,n,m);

	fill(mat_a,n,m);

	int **d_mat_a;
	int **dd_mat_a;
		

	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

	int i;

	d_mat_a = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat_a[0], sizeof(int) * m * n );
	hipMemcpy(d_mat_a[0], mat_a[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat_a[i]=(d_mat_a[i-1]+m);
	}	
	
	hipMalloc((void***)&dd_mat_a,size_row);
	hipMemcpy(dd_mat_a,d_mat_a,size_row,hipMemcpyHostToDevice);

	print(mat_a,n,m);
	printf("//////////////////\n");



	dim3 grid(N,N,1);
	dim3 blockNum(T,T,1);

	sum_matrix<<<grid,blockNum>>>(dd_mat_a,n,m);

	for(i=0;i<n;i++){
		hipMemcpy(mat_a[i],d_mat_a[i],size_col,hipMemcpyDeviceToHost);	
	}	

	printf("//////////////////\n");
	print(mat_a,n,m);


	return 0;
}