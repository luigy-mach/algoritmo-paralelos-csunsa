#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define TAM 10

#define N 2
#define T 6




__global__ 
void print_matrix(int** dd_mat_a, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){ 
		*(*(dd_mat_a+y)+x)=-3;
		printf("%d ",*(*(dd_mat_a+y)+x));
	}

}


__global__ 
void matrix_add(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int n, int m){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y<n && x<m ){ 
		*(*(dd_mat_c+y)+x)=*(*(dd_mat_a+y)+x)+*(*(dd_mat_b+y)+x);
	}

}

void create_host_matrix(int*** mat,int n, int m){
	*mat = (int** )malloc(sizeof(int*)*n);	
	(*mat)[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<n;i++){
		(*mat)[i] = (*mat)[0]+i*m;
	}
}



void fill_host_matrix(int** mat, int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%50;
	}
}


void fill_zero(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}

void create_matrices(int** &mat_a,int** &d_mat_a,int** &dd_mat_a,int n,int m)
{
	int i;

	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

	create_host_matrix(&mat_a,n,m);
	fill_host_matrix(mat_a,n,m);

	d_mat_a = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat_a[0], sizeof(int) * m * n );
	hipMemcpy(d_mat_a[0], mat_a[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat_a[i]=(d_mat_a[i-1]+m);
	}

	hipMalloc((void***)&dd_mat_a,size_row);
	hipMemcpy(dd_mat_a,d_mat_a,size_row,hipMemcpyHostToDevice);
	
}



int main(){


	int n = TAM;
	int m = TAM;

	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	int i;
	int size_row = sizeof(int*) * n;
	int size_col = sizeof(int ) * m;

    create_matrices(mat_a,d_mat_a,dd_mat_a,n,m);
    create_matrices(mat_b,d_mat_b,dd_mat_b,n,m);
    create_matrices(mat_c,d_mat_c,dd_mat_c,n,m);

    printf("Matrix A\n");
    print(mat_a,n,m);
    printf("\n");

    printf("Matrix B\n");
    print(mat_b,n,m);
    printf("\n");

    printf("Matrix C\n");
    print(mat_c,n,m);
    printf("\n");
    printf(\n");
    



	dim3 grid(N,N,1);
	dim3 blockNum(T,T,1);

	matrix_add<<<grid,blockNum>>>(dd_mat_a,dd_mat_b,dd_mat_c,n,m);

	for(i=0;i<n;i++){
		hipMemcpy(mat_c[i],d_mat_c[i],size_col,hipMemcpyDeviceToHost);	
	}	

	printf("//////////////////\n");
	print(mat_c,n,m);


	return 0;
}