#include "hip/hip_runtime.h"
// Luigy Machaca Arcana
// Computer science - Arequipa, Perú  2017


#include <stdlib.h>
#include <stdio.h>


#define WIDTH_TILE 32


__global__ void matrix_mult_shared(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	
	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int value = 0;

	int row = by*WIDTH_TILE + ty;
	int col = bx*WIDTH_TILE + tx;	

	int width = n_cols_a; //n_cols_a == n_rows_b

	int k;
	for( k=0 ; k<(int)(width-1+WIDTH_TILE)/(int)WIDTH_TILE ; ++k ){
		if (k*WIDTH_TILE+tx < n_cols_a && row < n_rows_a){
			Mds[ty][tx] = dd_mat_a[row][k*WIDTH_TILE+tx];
		}
        else{
			Mds[ty][tx] = 0;
        }

        if (k*WIDTH_TILE+ty < n_rows_b && col < n_cols_b){
			Nds[ty][tx] = dd_mat_b[k*WIDTH_TILE+ty][col];
        }
        else{
			Nds[ty][tx] = 0;
        }

		__syncthreads();
		int m;
		for(m=0 ; m<WIDTH_TILE ; ++m){
			value += Mds[ty][m]*Nds[m][tx];
		}
		__syncthreads();

	}

	if(row<n_rows_c && col<n_cols_c){
		dd_mat_c[row][col]=value;
	}
	

}


__global__ 
void matrix_mult(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){
	int value=0;


	int tx=threadIdx.x;
	int ty=threadIdx.y;


	int x = tx + blockIdx.x*blockDim.x;
	int y = ty + blockIdx.y*blockDim.y;

	if( y<n_rows_c && x<n_cols_c ){
		int i;
		for(i=0 ; i<n_cols_a ; i++){
			value += dd_mat_a[y][i] * dd_mat_b[i][x];
		}
		dd_mat_c[y][x]=value;
	} 
}



void fill(int** mat, int n, int m){
    srand(time(0));
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%3+1;
			//mat[i][j] = 1;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}




void create(int**& mat, int**& d_mat, int**& dd_mat, int n_rows, int n_cols, int fillValue=-1){
	
	int i;
	mat 	= (int** )malloc(sizeof(int*) * n_rows 			);	
	mat[0] 	= (int*  )malloc(sizeof(int ) * n_rows * n_cols );	

	for( i=1 ; i<n_rows ; i++ ){
		mat[i] = mat[i-1]+n_cols;
	}
	if(fillValue==-1){
		fill(mat,n_rows,n_cols);	
	}
	else{
		fill_value(mat,n_rows,n_cols,fillValue);
	}

	int size_row = sizeof(int*) * n_rows;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * n_rows * n_cols );
	hipMemcpy(  d_mat[0], mat[0], sizeof(int) * n_rows * n_cols ,hipMemcpyHostToDevice);

	for( i=1 ; i<n_rows ; i++ ){
		d_mat[i] = (d_mat[0]+i*n_cols);
	}	
	
	hipMalloc((void***)& dd_mat, size_row );
	hipMemcpy( dd_mat, d_mat, size_row, hipMemcpyHostToDevice );

}




int main(int argc, char *argv[]){


	int n_rows_a = 3;
	int n_cols_a = 5;

	int n_rows_b = 5;
	int n_cols_b = 7;

	int n_rows_c = n_rows_a;
	int n_cols_c = n_cols_b;


	if(n_cols_a!=n_rows_b){
		printf("error n_cols_a!=n_rows_b");
		return 0;
	}

	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	create( mat_a, d_mat_a, dd_mat_a, n_rows_a, n_cols_a	);
	create( mat_b, d_mat_b, dd_mat_b, n_rows_b, n_cols_b 	);
	create( mat_c, d_mat_c, dd_mat_c, n_rows_c, n_cols_c, 0	);


	/////////////////////////////////////////

	dim3 blockNum(WIDTH_TILE,WIDTH_TILE,1);
	dim3 grid((int)(n_cols_c-1+blockNum.x)/blockNum.x,(int)(n_rows_c-1+blockNum.y)/blockNum.y,1);
	printf("ty: %d, tx: %d\n",(int)(n_rows_c-1+blockNum.y)/blockNum.y, (int)(n_cols_c-1+blockNum.x)/blockNum.x);
	printf("grid_row: %d, grid_col: %d\n",grid.x , grid.y );

	////////////////////////////////////////////////////


	matrix_mult_shared<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//matrix_mult_shared_mejorado<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);

	//matrix_mult<<<grid,blockNum>>>(dd_mat_a, n_rows_a, n_cols_a, dd_mat_b, n_rows_b, n_cols_b, dd_mat_c, n_rows_c, n_cols_c);
	

    /////////////////////////////////////////////////////

	hipMemcpy(mat_c[0],d_mat_c[0],sizeof(int)*n_rows_c*n_cols_c,hipMemcpyDeviceToHost);		
	
	
	printf("//////////////////\n");
	printf("//////////////////\n");
	print(mat_a,n_rows_a,n_cols_a);
	printf("//////////////////\n");
	print(mat_b,n_rows_b,n_cols_b);

	printf("//////////////////\n");
	print(mat_c,n_rows_c,n_cols_c);
	
	
	


	hipFree(dd_mat_a);
	hipFree(dd_mat_b);
	hipFree(dd_mat_c);
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
  	
  	free(mat_a);
  	free(mat_b);
  	free(mat_c);
  	

	return 0;
}