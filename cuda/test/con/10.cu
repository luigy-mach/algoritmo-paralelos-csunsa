#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define WIDTH_TILE 70
#define TPB 32




__global__ 
void matrix_mult(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int n, int m){
	int value=0;
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if( y>n && x>m ) return;
	int i;
	for(i=0;i<m;i++){
		value += *(*(dd_mat_a)+y*m+i)  * *(*(dd_mat_b)+i*m+x);
	}

	*(*(dd_mat_c)+y*m+x)=value;


}



__global__ 
void matrix_mult_shared(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int width){

	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx=blockIdx.x;
	int by=blockIdx.y;

	int tx=threadIdx.x;
	int ty=threadIdx.y;



	int value=0;

	int fil = by*WIDTH_TILE+ty;
	int col = bx*WIDTH_TILE+tx;

	int m;
	int k;
	for(k=0 ; k<width/WIDTH_TILE ; k++){
		Mds[ty][tx]=dd_mat_a[fil][k+WIDTH_TILE+tx];
		Nds[ty][tx]=dd_mat_b[k*WIDTH_TILE+ty][col];
		__syncthreads();

		for(m=0;m<WIDTH_TILE;m++){
			value+=Mds[ty][m]*Nds[m][tx];
		}
		__syncthreads();

	}
	dd_mat_c[fil][col]=value;
}



void fill(int** mat, int n, int m){
	srand(time(0));
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%3+1;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}




void create(int**& mat, int**& d_mat, int**& dd_mat, int n, int m, int fillValue=-1){
	int i;

	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	for(i=1;i<n;i++){
		mat[i] = mat[i-1]+m;
	}
	if(fillValue==-1){
		fill(mat,n,m);	
	}
	else{
		fill_value(mat,n,m,fillValue);
	}

	int size_row = sizeof(int*) * n;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * m * n );
	hipMemcpy(d_mat[0], mat[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat[i]=(d_mat[0]+i*m);
	}	
	
	hipMalloc((void***)&dd_mat,size_row);
	hipMemcpy(dd_mat,d_mat,size_row,hipMemcpyHostToDevice);


}




int main(){

	int tam = 10000;
	///////////////////////////////
	int n = tam;
	int m = tam;



	int i;
	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	create(mat_a,d_mat_a,dd_mat_a,n,m);
	create(mat_b,d_mat_b,dd_mat_b,n,m);
	create(mat_c,d_mat_c,dd_mat_c,n,m,0);


	int size_col = sizeof(int ) * m;

	//print(mat_a,n,m);
	printf("//////////////////\n");
	//print(mat_b,n,m);
	printf("//////////////////\n");
	//print(mat_c,n,m);
	//printf("//////////////////\n");
	//printf("//////////////////\n");




	dim3 blockNum(TPB,TPB);
	dim3 grid((n + TPB-1)/blockNum.x,(m + TPB-1)/blockNum.y);



	/////////////////////////////////////////
	float time;
	hipEvent_t my_start,my_stop;
	hipEventCreate(&my_start);
	hipEventCreate(&my_stop);

	///////////////////////////////////////// TIME
    hipEventRecord(my_start,0);
	matrix_mult_shared<<<grid,blockNum>>>(dd_mat_a,dd_mat_b,dd_mat_c,n);
	//matrix_mult<<<grid,blockNum>>>(dd_mat_a,dd_mat_b,dd_mat_c,n);
    hipEventRecord(my_stop,0);
    hipEventSynchronize(my_stop);
    /////////////////////////////////////////////////////

	for(i=0;i<n;i++){
		hipMemcpy(mat_c[i],d_mat_c[i],size_col,hipMemcpyDeviceToHost);	
	}	

    hipEventElapsedTime(&time,my_start,my_stop);
	printf("time %dX%d , tam %d : %.25f \n",WIDTH_TILE,WIDTH_TILE,tam,time);

	//printf("//////////////////\n");
	//printf("//////////////////\n");
	//print(mat_a,n,m);
	//printf("//////////////////\n");
	//print(mat_b,n,m);
	printf("//////////////////\n");
	//print(mat_c,n,m);



	hipFree(dd_mat_a);
	hipFree(dd_mat_b);
	hipFree(dd_mat_c);
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
  	
  	free(mat_a);
  	free(mat_b);
  	free(mat_c);
  	

	return 0;
}
