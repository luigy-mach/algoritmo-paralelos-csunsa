#include "hip/hip_runtime.h"
// Luigy Machaca Arcana
// Computer science - Arequipa, Perú  2017

/*
CUDA Device Query...
There are 1 CUDA devices.

CUDA Device #0
Major revision number:         3
Minor revision number:         0
Name:                          GeForce GT 750M
Total global memory:           2098724864
Total shared memory per block: 49152
Total registers per block:     65536
Warp size:                     32
Maximum memory pitch:          2147483647
Maximum threads per block:     1024
Maximum dimension 0 of block:  1024
Maximum dimension 1 of block:  1024
Maximum dimension 2 of block:  64
Maximum dimension 0 of grid:   2147483647
Maximum dimension 1 of grid:   65535
Maximum dimension 2 of grid:   65535
Clock rate:                    967000
Total constant memory:         65536
Texture alignment:             512
Concurrent copy and execution: Yes
Number of multiprocessors:     2
Kernel execution timeout:      Yes

Press any key to exit...


*/


#include <stdlib.h>
#include <stdio.h>


#define WIDTH_TILE 32




__global__ void matrix_mult_shared(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	
	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx=blockIdx.x;
	int by=blockIdx.y;

	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int value = 0;

	int row = by*WIDTH_TILE + ty;
	int col = bx*WIDTH_TILE + tx;	

	int width = n_cols_a; //n_cols_a == n_rows_b

	int k;
	for(k=0 ; k<(int)(width-1+WIDTH_TILE)/(int)WIDTH_TILE ; ++k){
		if (k*WIDTH_TILE+tx < n_cols_a && row < n_rows_a){
			Mds[ty][tx] = dd_mat_a[row][k*WIDTH_TILE+tx];
		}
        else{
			Mds[ty][tx] = 0;
        }

        if (k*WIDTH_TILE+ty < n_rows_b && col < n_cols_b){
			Nds[ty][tx] = dd_mat_b[k*WIDTH_TILE+ty][col];
        }
        else{
			Nds[ty][tx] = 0;
        }

		__syncthreads();
		int m;
		for(m=0 ; m<WIDTH_TILE ; ++m){
			value += Mds[ty][m]*Nds[m][tx];
		}
		__syncthreads();

	}

	if(row<n_rows_c && col<n_cols_c){
		dd_mat_c[row][col]=value;
	}
	

}



__global__ void matrix_mult_shared_mejorado(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){

	
	__shared__ int Mds[WIDTH_TILE][WIDTH_TILE];
	__shared__ int Nds[WIDTH_TILE][WIDTH_TILE];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Row = by*WIDTH_TILE + ty;
	int Col1  = (bx*2)*WIDTH_TILE + tx;
	int Col2 = (bx*2+1)*WIDTH_TILE + tx;

	int value1 = 0;
	int value2 = 0;
	
	int k = 0;
	int prefM  = dd_mat_a[Row][k*WIDTH_TILE + tx];
	int prefN  = dd_mat_b[k*WIDTH_TILE + ty][Col1];
	int prefN2 = dd_mat_b[k*WIDTH_TILE + ty][Col2];
		

	Mds[ty][tx] = prefM;
	Nds[ty][tx] = prefN;
	__syncthreads();
	

	int width = n_cols_a; //n_cols_a == n_rows_b

	for(int m = 0; m < (int)(width-1+WIDTH_TILE)/(int)WIDTH_TILE ; ++m){				
		
		prefM = dd_mat_a[Row][m*WIDTH_TILE + tx];
		prefN = dd_mat_b[(m*WIDTH_TILE + ty)][Col1];
		
		for(int k = 0; k<WIDTH_TILE; k++){
			value1 += Mds[ty][k] * Nds[k][tx];
		}		
		__syncthreads();
		
		Nds[ty][tx] = prefN2;
		
		__syncthreads();
		
		prefN2 = dd_mat_b[(m*WIDTH_TILE + ty)][Col2];
		
		for(int k = 0; k < WIDTH_TILE; k++){
			value2 += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
		
		
		Mds[ty][tx] = prefM;
		Nds[ty][tx] = prefN;
		//__syncthreads();		
		
	}


	if( Row<n_rows_c && Col1<n_cols_c ){
		dd_mat_c[Row][Col1] = value1;
	}

	if( Row<n_rows_c && Col2<n_cols_c ){
		dd_mat_c[Row][Col2] = value2;
	}

}





__global__ 
void matrix_mult(int** dd_mat_a, int n_rows_a, int n_cols_a ,int** dd_mat_b, int n_rows_b, int n_cols_b, int** dd_mat_c, int n_rows_c, int n_cols_c){
	int value=0;


	int tx=threadIdx.x;
	int ty=threadIdx.y;


	int x = tx + blockIdx.x*blockDim.x;
	int y = ty + blockIdx.y*blockDim.y;

	if( y<n_rows_c && x<n_cols_c ){
		int i;
		for(i=0 ; i<n_cols_a ; i++){
			value += dd_mat_a[y][i] * dd_mat_b[i][x];
		}
		dd_mat_c[y][x]=value;
	} 
}



void fill(int** mat, int n, int m){
    srand(time(0));
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			mat[i][j] = rand()%3+1;
			//mat[i][j] = 1;
	}
}


void fill_value(int** mat,int n, int m, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			mat[i][j] = value;
}


void print(int** mat,int n, int m){
	int i,j; 
	for(i=0; i<n ;i++){
		for(j=0; j<m ;j++)
			printf("%d ",mat[i][j]);
		printf("\n");
	}
}




void create(int**& mat, int**& d_mat, int**& dd_mat, int n, int m, int fillValue=-1){
	int i;

	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	for(i=1;i<n;i++){
		mat[i] = mat[i-1]+m;
	}
	if(fillValue==-1){
		fill(mat,n,m);	
	}
	else{
		fill_value(mat,n,m,fillValue);
	}

	int size_row = sizeof(int*) * n;

	d_mat = (int**) malloc(size_row);
	hipMalloc((void**)& d_mat[0], sizeof(int) * m * n );
	hipMemcpy(d_mat[0], mat[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);
	for(i=1;i<n;i++){
		d_mat[i]=(d_mat[0]+i*m);
	}	
	
	hipMalloc((void***)&dd_mat,size_row);
	hipMemcpy(dd_mat,d_mat,size_row,hipMemcpyHostToDevice);


}




int main(int argc, char *argv[]){

	//int tam = 512;
	int tam = atoi(argv[1]);


	int n = tam;
	int m = tam;
	int p = tam;
	int q = tam;


	if(m!=p){
		printf("error m!=p");
		return 0;
	}

	int** mat_a; int** d_mat_a;	 int** dd_mat_a;	
	int** mat_b; int** d_mat_b;	 int** dd_mat_b;	
	int** mat_c; int** d_mat_c;	 int** dd_mat_c;	

	create(mat_a,d_mat_a,dd_mat_a,n,m);
	create(mat_b,d_mat_b,dd_mat_b,n,m);
	create(mat_c,d_mat_c,dd_mat_c,n,m,0);



	/////////////////////////////////////////
	float time1,time2,time3;
	hipEvent_t my_start,my_stop;
	hipEventCreate(&my_start);
	hipEventCreate(&my_stop);


	dim3 blockNum(WIDTH_TILE,WIDTH_TILE,1);
	dim3 grid((int)(n-1+blockNum.x)/blockNum.x,(int)(q-1+blockNum.y)/blockNum.y,1);
	//printf("tx: %d,ty: %d\n",(int)(n-1+blockNum.x)/blockNum.x,(int)(q-1+blockNum.y)/blockNum.y);
	//printf("grid_row: %d, grid_col: %d\n",grid.y , grid.x );

	///////////////////////////////////////// TIME1
    hipEventRecord(my_start,0);

	matrix_mult_shared<<<grid,blockNum>>>(dd_mat_a,n,m,dd_mat_b,p,q,dd_mat_c,n,q);
	
    hipEventRecord(my_stop,0);
    hipEventSynchronize(my_stop);
    hipEventElapsedTime(&time1,my_start,my_stop);
    /////////////////////////////////////////////////////


    ///////////////////////////////////////// TIME2
    hipEventRecord(my_start,0);

	matrix_mult_shared_mejorado<<<grid,blockNum>>>(dd_mat_a,n,m,dd_mat_b,p,q,dd_mat_c,n,q);

    hipEventRecord(my_stop,0);
    hipEventSynchronize(my_stop);
    hipEventElapsedTime(&time2,my_start,my_stop);
    /////////////////////////////////////////////////////


    ///////////////////////////////////////// TIME3
    hipEventRecord(my_start,0);

	matrix_mult<<<grid,blockNum>>>(dd_mat_a,n,m,dd_mat_b,p,q,dd_mat_c,n,q);
	
    hipEventRecord(my_stop,0);
    hipEventSynchronize(my_stop);
    hipEventElapsedTime(&time3,my_start,my_stop);
    /////////////////////////////////////////////////////



	//printf("time1 %dX%d , tam %d : %.25f \n",WIDTH_TILE,WIDTH_TILE,tam,time1/1000);
	//printf("time2 %dX%d , tam %d : %.25f \n",WIDTH_TILE,WIDTH_TILE,tam,time2/1000);
	//printf("time2 %dX%d , tam %d : %.25f \n",WIDTH_TILE,WIDTH_TILE,tam,time3/1000);

	hipMemcpy(mat_c[0],d_mat_c[0],sizeof(int)*n*q,hipMemcpyDeviceToHost);		

	printf("%.25f,%.25f,%.25f",time1/1000,time2/1000,time3/1000);

	
	/*
	printf("//////////////////\n");
	printf("//////////////////\n");
	print(mat_a,n,m);
	printf("//////////////////\n");
	print(mat_b,p,q);

	printf("//////////////////\n");
	print(mat_c,n,q);
	*/
	
	


	hipFree(dd_mat_a);
	hipFree(dd_mat_b);
	hipFree(dd_mat_c);
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);
  	
  	free(mat_a);
  	free(mat_b);
  	free(mat_c);
  	

	return 0;
}